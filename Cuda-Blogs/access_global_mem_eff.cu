// https://developer.nvidia.com/blog/how-access-global-memory-efficiently-cuda-c-kernels/

#include<stdio.h>
#include<stdlib.h>
#include<assert.h>
#include<hip/hip_runtime.h>

//This code can run both offset and stride kernels in either single (default) or double precision by passing the “fp64” command line option. 
// Each kernel takes two arguments, an input array and an integer representing the offset or stride used to access the elements of the array. The kernels are called in loops over a range of offsets and strides.



// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

template <typename T>
__global__ void offset(T* a, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i] + 1;
}

template <typename T>
__global__ void stride(T* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i] + 1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  int blockSize = 256;
  float ms;

  T *d_a;
  hipEvent_t startEvent, stopEvent;
    
  int n = nMB*1024*1024/sizeof(T);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Offset, Bandwidth (GB/s):\n");
  
  offset<<<n/blockSize, blockSize>>>(d_a, 0); // warm up

  for (int i = 0; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    offset<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  printf("\n");
  printf("Stride, Bandwidth (GB/s):\n");

  stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
  for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int nMB = 4;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  checkCuda( hipSetDevice(deviceId) );
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");
  
  if(bFp64){
    runTest<double>(deviceId, nMB);    
  } 
  else{
    runTest<float>(deviceId, nMB);
  } 
}