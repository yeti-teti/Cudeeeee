#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void kernel(uint *A, uint *B, int row) {
  auto x = threadIdx.x / 4;
  auto y = threadIdx.x % 4;
  A[x * row + y] = x;
  B[x * row + y] = y;
}

int main(int argc, char **argv) {
  uint *Xs, *Ys;
  uint *Xs_d, *Ys_d;

  uint SIZE = 4;

  Xs = (uint *)malloc(SIZE * SIZE * sizeof(uint));
  Ys = (uint *)malloc(SIZE * SIZE * sizeof(uint));

  hipMalloc((void **)&Xs_d, SIZE * SIZE * sizeof(uint));
  hipMalloc((void **)&Ys_d, SIZE * SIZE * sizeof(uint));

  dim3 grid_size(1, 1, 1);
  dim3 block_size(4 * 4);

  kernel<<<grid_size, block_size>>>(Xs_d, Ys_d, 4);

  hipMemcpy(Xs, Xs_d, SIZE * SIZE * sizeof(uint), hipMemcpyDeviceToHost);
  hipMemcpy(Ys, Ys_d, SIZE * SIZE * sizeof(uint), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for (int row = 0; row < SIZE; ++row) {
    for (int col = 0; col < SIZE; ++col) {
      std::cout << "[" << Xs[row * SIZE + col] << "|" << Ys[row * SIZE + col]
                << "] ";
    }
    std::cout << "\n";
  }

  hipFree(Xs_d);
  hipFree(Ys_d);
  free(Xs);
  free(Ys);
}